#include "hip/hip_runtime.h"
#include <stdio.h>
#include <shmem.h>

#define N (1<<20)

__global__
void add(int n, int *x, int *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(int argc, char *argv[]) {
  int exitcode = 0;
  int *a, *b;

  shmem_init();
  int me = shmem_my_pe();
  int npes = shmem_n_pes();

  a = (int *) shmem_malloc(N * sizeof(int));
  hipMallocManaged(&b, N * sizeof(int));

  for (int i = 0; i < N; i++) {
    a[i] = 1; b[i] = 2;
  }

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  add<<<numBlocks, blockSize>>>(N, a, b);

  hipDeviceSynchronize();

  shmem_barrier_all();
  shmem_int_put(a, b, N, (me + 1) % npes);
  shmem_barrier_all();

  for (int i = 0; i < N; i++) {
    if (a[i] != b[i]) {
      fprintf(stderr, "[PE %d] ERROR: expected %d, found %d\n", 
                      me, b[i], a[i]);
      exitcode = 1;
    }
  }

  shmem_free(a);
  hipFree(b);
  shmem_finalize();

  return exitcode;
}
