#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <shmem.h>

#define MIN_MSG_SZ (1<<0)
#define MAX_MSG_SZ (1<<20)
#define NUM_ITER 100
#define WARMUP 10

__global__
void init_data(int n, int *x, int *y, int value)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    x[i] = value * value + value;
    y[i] = value * value + value + 1;
  }
}

#ifdef USE_RDTSC
static inline uint64_t rdtsc()
{
  unsigned int hi, lo;
  __asm__ volatile("rdtsc" : "=a" (lo), "=d" (hi));
  return ((uint64_t)hi << 32) | lo;
}

static inline double GetGHzFreq()
{
  unsigned long long startTime = rdtsc();
  sleep(1);
  unsigned long long endTime = rdtsc();
  return (double)(endTime - startTime) / 1.0e9;
}
#else
double wtime(void)
{
  double wtime = 0.0;

#ifdef CLOCK_MONOTONIC
  struct timespec tv;
  clock_gettime(CLOCK_MONOTONIC, &tv);
  wtime = tv.tv_sec * 1e6;
  wtime += (double)tv.tv_nsec / 1000.0;
#else
  struct timeval tv;
  gettimeofday(&tv, NULL);
  wtime = tv.tv_sec * 1e6;
  wtime += (double)tv.tv_usec;
#endif
  return wtime;
}
#endif

double get_time() {
#ifdef USE_RDTSC
  return rdtsc();
#else
  return wtime();
#endif
}

int main(int argc, char *argv[]) {
  int exitcode = 0;
  int *a, *b;
  int msg_size, i;
  double start_time = 0.0, end_time = 0.0;
#ifdef PROFILE
  double profile_start = 0.0, profile_time = 0.0;
#endif

  shmem_init();
  int me = shmem_my_pe();
  int npes = shmem_n_pes();

  if (me == 0) {
#ifdef USE_DEVICE
    fprintf(stderr, "Device initialization test\n");
#else
    fprintf(stderr, "Host initialization test\n");
#endif
  }

  a = (int *) shmem_malloc(MAX_MSG_SZ * sizeof(int));
  hipMallocManaged(&b, MAX_MSG_SZ * sizeof(int));

#ifdef USE_RDTSC
  double freq = GetGHzFreq();
#endif

  for (msg_size = MIN_MSG_SZ; msg_size <= MAX_MSG_SZ; msg_size *= 2) {
#ifdef PROFILE
    profile_time = 0.0;
#endif
    for (i = 0; i < (NUM_ITER + WARMUP); i++) {
      shmem_barrier_all();
      if (i == WARMUP && me == 0) start_time = get_time();
#ifdef USE_DEVICE
      int block_size = 256;
      int num_blocks = (msg_size + block_size - 1) / block_size;
      init_data<<<num_blocks, block_size>>>(msg_size, a, b, i);
#ifdef PROFILE
      if (me == 0) profile_start = get_time();
#endif
      hipDeviceSynchronize();
#ifdef PROFILE
      if (me == 0) profile_time += (get_time() - profile_start);
#endif
#else
      int j;
      for (j = 0; j < msg_size; j++) {
        a[j] = i * i + i;
        b[j] = i * i + i + 1;
      }
#endif
      shmem_barrier_all();
      shmem_int_put(a, b, msg_size, (me + 1) % npes);
    }
    if (me == 0) end_time = get_time();
    shmem_barrier_all();

#ifdef USE_RDTSC
    if (me == 0) { 
      fprintf(stderr, "%10d%10s%10.2f", msg_size, " ", 
              (double)((end_time - start_time) / ((double) NUM_ITER * freq * 1.0e3)));
#ifdef PROFILE
      fprintf(stderr, "%10s%10.2f\n", " ", 
              (double)(profile_time / ((double) NUM_ITER * freq * 1.0e3)));
#else
      fprintf(stderr, "\n");
#endif
    }
#else
    if (me == 0) { 
      fprintf(stderr, "%10d%10s%10.2f", msg_size, " ", 
              (double)((end_time - start_time) / NUM_ITER));
#ifdef PROFILE
      fprintf(stderr, "%10s%10.2f\n", " ", (double)(profile_time / NUM_ITER));
#else
      fprintf(stderr, "\n");
#endif
    }
#endif

    for (i = 0; i < msg_size; i++) {
      if (a[i] != b[i]) {
        fprintf(stderr, "[PE %d] ERROR: expected %d, found %d\n",
                        me, b[i], a[i]);
        exitcode = 1;
        break;
      }
    }

    if (exitcode) break;
  }

  shmem_free(a);
  hipFree(b);
  shmem_finalize();

  return exitcode;
}
